//==--- ripple/core/tests/boundary_device.cu -------------------- -*- C++ -*- ---==//
//            
//                                Ripple
// 
//                      Copyright (c) 2019 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  boundary_device.cu
/// \brief This file runs device boundary tests.
//
//==------------------------------------------------------------------------==//

#include "boundary/boundary_tests_device.hpp"
#include <gtest/gtest.h>

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
