#include "hip/hip_runtime.h"
//==--- ripple/benchmarks/reinitialization.cu -------------- -*- C++ -*- ---==//
//
//                                Ripple
//
//                      Copyright (c) 2019, 2020 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  reinitialization.cu
/// \brief This file defines a benchmark for levelset reinitialization.
//
//==------------------------------------------------------------------------==//

#include "fim_solver.hpp"
#include <ripple/core/boundary/fo_extrap_loader.hpp>
#include <ripple/core/container/tensor.hpp>
#include <ripple/core/execution/executor.hpp>
#include <ripple/core/utility/timer.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>

/*
 * This benchmarks reinitializes levelset data using the fast iterative method.
 * Usage is:
 *    ./reinitializtion <elements per dim> <bandwidth>
 */

/** Number of dimensions for the solver. */
constexpr size_t dims = 2;

using Real    = float;
using Element = LevelsetElement<Real, ripple::StridedView>;
using Tensor  = ripple::Tensor<Element, dims>;

/**
 * Makes a tensor with the given number of elements per dimension and padding
 * elements.
 * \param elements The number of elements per dimension.
 * \param padding  The number of padding elements per side of the dimension.
 */
template <size_t Dims>
auto make_tensor(size_t elements, uint32_t padding = 0) noexcept {
  if constexpr (Dims == 1) {
    return ripple::Tensor<Element, 1>{{1}, padding, elements};
  } else if constexpr (Dims == 2) {
    return ripple::Tensor<Element, 2>{{1, 1}, padding, elements, elements};
  } else if constexpr (Dims == 3) {
    return ripple::Tensor<Element, 3>{
      {1, 1, 1}, padding, elements, elements, elements};
  }
}

int main(int argc, char** argv) {
  size_t elements = 100;
  size_t padding  = 1;
  size_t iters    = 20;
  if (argc > 1) {
    elements = std::atol(argv[1]);
  }
  if (argc > 2) {
    iters = std::atol(argv[2]);
  }

  /*
   * NOTE: NVCC does *not* allow generic extended lambdas, so we need the type
   *       of the iterator if we want to pass lanmbdas to the methods to create
   *       the graph.
   *
   *       This is restrictive in that we need different iterators to global
   *       and shared data, and hence if we use ripple::in_shared() on the
   *       tensor data then we *also* need to change the iterator type, which is
   *       annoying.
   *
   *       We can get around this by defining the lamdas as functors with
   *       generic templates, i,e
   *
   *          template <typename It>
   *          ripple_host_device auto operator()(It&& it) const -> void {}
   *
   *       But for a simple case like this, the lamdas are nice.
   */
  auto data            = make_tensor<dims>(elements, padding);
  using Traits         = ripple::tensor_traits_t<decltype(data)>;
  using Iterator       = typename Traits::Iterator;
  using SharedIterator = typename Traits::SharedIterator;

  ripple::Graph init(ripple::ExecutionKind::gpu);
  init
    .split(
      [] ripple_host_device(Iterator it) {
        if (it.first_in_global_space()) {
          it->value() = 0;
          it->state() = State::source;
          return;
        }
        it->value() = std::numeric_limits<Real>::max();
        it->state() = State::updatable;
      },
      data)
    .then_split(
      [] ripple_host_device(Iterator it) {
        ripple::load_boundary(it, ripple::FOExtrapLoader());
      },
      data);
  ripple::execute(init);
  ripple::fence();

  ripple::Graph solve;
  Real          dh = 0.1;
  solve.split(
    [] ripple_host_device(SharedIterator it, Real dh, size_t iters) {
      constexpr auto fim_solve = FimSolver();
      fim_solve(it, dh, iters);
    },
    ripple::in_shared(data),
    dh,
    iters);

  ripple::Timer timer;
  ripple::execute(solve);
  ripple::fence();

  double elapsed = timer.elapsed_msec();
  std::cout << "Size: " << elements << "x" << elements
            << " elements, Iters: " << iters << ", Time: " << elapsed
            << " ms\n";

  if (elements < 15) {
    for (size_t j = 0; j < elements; ++j) {
      for (size_t i = 0; i < elements; ++i) {
        printf("%5.5f ", data(i, j)->value());
      }
      printf("\n");
    }
  }

  return 0;
}