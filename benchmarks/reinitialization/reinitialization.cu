#include "hip/hip_runtime.h"
/**=--- ../benchmarks/reinitialization/reinitialization.cu - -*- C++ -*- ---==**
 *
 *                                  Ripple
 *
 *                      Copyright (c) 2019 - 2021 Rob Clucas.
 *
 *  This file is distributed under the MIT License. See LICENSE for details.
 *
 *==-------------------------------------------------------------------------==*
 *
 * \file  reinitialization.cu
 * \brief This file defines a benchmark for levelset reinitialization.
 *
 *==------------------------------------------------------------------------==*/

#include "fim_solver.hpp"
#include <ripple/container/tensor.hpp>
#include <ripple/execution/executor.hpp>
#include <ripple/padding/fo_extrap_loader.hpp>
#include <ripple/utility/timer.hpp>
#include <iostream>

/*
 * This benchmarks reinitializes levelset data using the fast iterative method.
 * Usage is:
 *    ./reinitializtion <elements per dim> <bandwidth>
 */

/** Number of dimensions for the solver. */
constexpr size_t dims = 2;

using Real = float;
using Elem = Element<Real, ripple::StridedView>;

/**
 * Makes a tensor with the given number of elements per dimension and padding
 * elements.
 * \param elements The number of elements per dimension.
 * \param padding  The number of padding elements per side of the dimension.
 */
template <size_t Dims>
auto make_tensor(
  size_t elements, uint32_t padding = 0, uint32_t partitions = 1) noexcept {
  if constexpr (Dims == 1) {
    return ripple::Tensor<Elem, 1>{{1}, padding, elements};
  } else if constexpr (Dims == 2) {
    return ripple::Tensor<Elem, 2>{
      {1, partitions}, padding, elements, elements};
  } else if constexpr (Dims == 3) {
    return ripple::Tensor<Elem, 3>{
      {1, partitions, 1}, padding, elements, elements, elements};
  }
}

/**
 * Initialization functor.
 */
struct Initializer {
  /**
   * Overload of operator() to call the initializer.
   * \param  it       The iterator to initialize the data for.
   * \tparam Iterator The type of the iterator.
   */
  template <typename Iterator>
  ripple_all auto operator()(Iterator it) const noexcept -> void {
    constexpr size_t source_loc = 5;
    bool             is_source  = true;
    ripple::unrolled_for<dims>([&](auto dim) {
      if (it.global_idx(dim) != source_loc) {
        is_source = false;
      }
    });

    if (is_source) {
      it->value() = 0;
      it->state() = State::source;
    } else {
      it->value() = std::numeric_limits<Real>::max();
      it->state() = State::updatable;
    }
  }
};

int main(int argc, char** argv) {
  size_t elements   = 10;
  size_t padding    = 6;
  size_t iters      = 2;
  size_t partitions = 2;
  size_t expansion  = 2;
  if (argc > 1) {
    elements = std::atol(argv[1]);
  }
  if (argc > 2) {
    iters = std::atol(argv[2]);
  }
  if (argc > 3) {
    expansion = std::atol(argv[3]);
  }
  if (argc > 4) {
    partitions = std::atol(argv[4]);
  }

  /*
   * NOTE: NVCC does *not* allow generic extended lambdas, so we need the type
   *       of the iterator if we want to pass lanmbdas to the methods to
   *       create the graph.
   *
   *       This is restrictive in that we need different iterators to global
   *       and shared data, and hence if we use ripple::in_shared() on the
   *       tensor data then we *also* need to change the iterator type, which
   *      is annoying.
   *
   *       We can get around this by defining the lamdas as functors with
   *       generic templates, i,e
   *
   *          template <typename It>
   *          ripple_all auto operator()(It&& it) const -> void {}
   *
   *       But for a simple case like this, the lamdas are nice.
   */
  auto data    = make_tensor<dims>(elements, padding, partitions);
  using Traits = ripple::tensor_traits_t<decltype(data)>;

  ripple::Graph init(ripple::ExecutionKind::gpu);
  /* First we initialize the data, so that the source node is set. We then
   * load the padding data for each partition, so that the values outside the
   * domain (in the padding) are valid and errors are not propogated into the
   * domain.
   */
  init.split(Initializer(), data)
    .then_split(ripple::LoadPadding(), data, ripple::FOExtrapLoader());
  ripple::execute(init);
  ripple::fence();

  ripple::Graph solve;
  Real          dh = 0.1;

  /* First we need to copy padding from the neighbour partition so that we
   * don't need to communicate during the computation, then we can execute the
   * solver, which will run for each partition. */
  solve.memcopy_padding(ripple::concurrent_padded_access(data))
    .then_split(
      FimSolver(),
      // ripple::expanded(data, expansion),
      // ripple::in_shared(data),
      data,
      ripple_move(dh),
      ripple_move(iters));

  ripple::Timer timer;
  ripple::execute(solve);
  ripple::barrier();

  double elapsed = timer.elapsed_msec();
  std::cout << "Size: " << elements << "x" << elements
            << " elements, Iters: " << iters << ", Time: " << elapsed
            << " ms\n";

  // For small sizes, print the result to test that it's working.
  if (elements < 30) {
    printf("      ");
    for (size_t i = 0; i < elements; ++i) {
      printf("%05lu ", i);
    }
    printf("\n");
    for (size_t j = 0; j < elements; ++j) {
      printf("%5lu ", j);
      for (size_t i = 0; i < elements; ++i) {
        if (data(i, j)->value() < 99.0f) {
          printf("%05.2f ", data(i, j)->value());
        } else {
          printf("----- ");
        }
      }
      printf("\n");
    }
  }

  return 0;
}